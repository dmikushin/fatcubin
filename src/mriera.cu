#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/mman.h>
#include <sys/stat.h>

#include "mriera.h"

using namespace std;

static unsigned long long _find_cubin_offset(ElfW(Shdr) header,
	void* start_ptr, unsigned long long offset, const char* name)
{
	return 0;
}

int main(int argc, char * argv[])
{
	if (argc != 3)
	{
		printf("%s <elf_filename> <kernel_name>\n", argv[0]);
		return 0;
	}

	const char* filename = argv[1];
	const char* kernel_name = argv[2];

	void* start_ptr = NULL;
	struct stat sb;
	size_t sz = 0;

	// Either Elf64_Ehdr or Elf32_Ehdr depending on architecture.
	ElfW(Ehdr) elf_header;
	ElfW(Shdr) header;

	cout << "opening elf file" << endl;
	FILE* file = fopen(filename, "rb");

	int fd = fileno(file);
	if (fd < 0)
	{
		printf("Could not open file for memory mapping, fd = %i\n", errno);
		exit(1);
	}

	cout << "getting file size" << endl;
	if (fstat(fd, &sb) == -1)					// To obtain file size
		printf("Could not find fstat");
	sz = sb.st_size;

	cout << "Mapping file to memory : " << sz << endl;
	start_ptr = mmap(NULL, sz, PROT_READ | PROT_WRITE, MAP_PRIVATE, fd, 0);

	//check if valid elf
	bool b = elf_is_elf64(file);
	fseek(file, 0, SEEK_SET);
	cout << "is ELF file : " << b << endl;
	if (b)
	{
		cout << "Found valid ELF file" << endl;
		//get ELF_Header
		b = elf64_get_elf_header(file, &elf_header);
		fseek(file, 0, SEEK_SET);

		if (b)
		{
			cout << "-Found valid ELF Header" << endl;
			b = elf64_get_section_header_by_name(file, (const Elf64_Ehdr *) &elf_header, ".nv_fatbin", &header);
			fseek(file, 0, SEEK_SET);

			if (b)
			{
				cout << "Found fatbin section" << endl;
				hipInit(0);
				// Get number of devices supporting CUDA
				int deviceCount = 0;
				hipGetDeviceCount(&deviceCount);

				if (deviceCount == 0)
				{
					printf("There is no device supporting CUDA.\n");
					exit (0);
				}
				else cout << "Number of device is "<< deviceCount << endl;

				// Get handle for device 0
				hipDevice_t cuDevice;
				hipDeviceGet(&cuDevice, 0);
				// Create context
				hipCtx_t cuContext;
				int ret = hipCtxCreate(&cuContext, 0, cuDevice);
				if (ret != hipSuccess)
					cout << "Could not create context on device 0" << endl;
				// Create module from binary file
				hipModule_t cuModule;
				cout << "sh_addr = " <<	header.sh_addr << endl;
				unsigned long long offset = header.sh_addr;
				
				unsigned long long cuOffset = _find_cubin_offset(header, start_ptr, offset, kernel_name);

				const void * fatbin = &((unsigned char *) start_ptr)[cuOffset];
				
				 cout << "fatbin = " << fatbin << endl;

				ret = hipModuleLoadFatBinary(&cuModule, fatbin);

				if (ret != hipSuccess)
				{
					cout << "Failed to load fatbin : " << filename << " : " << ret << endl;
				}

				hipFunction_t khw;
				ret = hipModuleGetFunction(&khw, cuModule, kernel_name);
				if (ret != hipSuccess)
				{
					cout << "Failed to get " << kernel_name << " from " << filename << " : " << ret << endl;
				}
				else ret = hipModuleLaunchKernel(khw, 1, 1, 1, 1, 1, 1, 0, 0, NULL, 0);

				if (ret != hipSuccess)
				{
					cout << "Failed to launch : " << kernel_name << endl;
				}

				ret = hipModuleUnload(cuModule);

				if (ret != hipSuccess)
				{
					cout << "Failed to unload self fatbin : " << filename << endl;
					return -1;
				}

				if (hipDeviceSynchronize() != hipSuccess)
				{
					printf ("Cuda call failed\n");
				}

				//unmap sutff
				munmap(start_ptr, sz);
				return 0;
			}
		}

	}

	fclose(file);

	return 0;
}

