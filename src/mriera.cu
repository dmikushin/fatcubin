#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/mman.h>
#include <sys/stat.h>

#include "mriera.h"

using namespace std;

static unsigned long long _find_cubin_offset(ElfW(Shdr) header,
	void* start_ptr, unsigned long long offset, const char* name)
{
	return 0;
}

int main(int argc, char * argv[])
{
	void* start_ptr = NULL;
	struct stat sb;
	size_t sz = 0;

	//read_elf_header(argv[0]);
	// Either Elf64_Ehdr or Elf32_Ehdr depending on architecture.
	ElfW(Ehdr) elf_header;
	ElfW(Shdr) header;

	cout << "opening elf file" << endl;
	FILE* file = fopen(argv[0], "rb");

	int fd = fileno(file);
	if (fd < 0)
	{
		printf("Could not open file for memory mapping, fd = %i\n", errno);
		exit(1);
	}

	cout << "getting file size" << endl;
	if (fstat(fd, &sb) == -1)					// To obtain file size
		printf("Could not find fstat");
	sz = sb.st_size;

	cout << "Mapping file to memory : " << sz << endl;
	start_ptr = mmap(NULL, sz, PROT_READ | PROT_WRITE, MAP_PRIVATE, fd, 0);

	//check if valid elf
	bool b = elf_is_elf64(file);
	fseek(file, 0, SEEK_SET);
	cout << "is ELF file : " << b << endl;
	if (b)
	{
		cout << "Found valid ELF file" << endl;
		//get ELF_Header
		b = elf64_get_elf_header(file, &elf_header);
		fseek(file, 0, SEEK_SET);

		if (b)
		{
			cout << "-Found valid ELF Header" << endl;
			b = elf64_get_section_header_by_name(file, (const Elf64_Ehdr *) &elf_header, ".nv_fatbin", &header);
			fseek(file, 0, SEEK_SET);

			if (b)
			{
				cout << "Found fatbin section" << endl;
				hipInit(0);
				// Get number of devices supporting CUDA
				int deviceCount = 0;
				hipGetDeviceCount(&deviceCount);

				if (deviceCount == 0)
				{
					printf("There is no device supporting CUDA.\n");
					exit (0);
				}
				else cout << "Number of device is "<< deviceCount << endl;

				// Get handle for device 0
				hipDevice_t cuDevice;
				hipDeviceGet(&cuDevice, 0);
				// Create context
				hipCtx_t cuContext;
				int ret = hipCtxCreate(&cuContext, 0, cuDevice);
				if (ret != hipSuccess)
					cout << "Could not create context on device 0" << endl;
				// Create module from binary file
				hipModule_t cuModule;
				cout << "sh_addr = " <<	header.sh_addr << endl;
				unsigned long long offset = header.sh_addr;
				
				unsigned long long cuOffset = _find_cubin_offset(header, start_ptr, offset, "_Z11hello_worldv");

				const void * fatbin = &((unsigned char *) start_ptr)[cuOffset];
				
				 cout << "fat bin = " << fatbin << endl;

				ret = hipModuleLoadFatBinary(&cuModule, fatbin);

				if (ret != hipSuccess)
				{
					cout << "Failed to load self fatbin : " << argv[0] << " : " << ret<< endl;
				}

				hipFunction_t khw;
				//ret = hipModuleGetFunction(&khw, cuModule, "hello_world");
				ret = hipModuleGetFunction(&khw, cuModule, "_Z11hello_worldv");
				if (ret != hipSuccess)
				{
					cout << "Failed to get hello_world from " << argv[0] << " : " << ret <<	endl;
				}
				else ret = hipModuleLaunchKernel(khw, 1, 1, 1, 1, 1, 1, 0, 0, NULL, 0);

				if (ret != hipSuccess)
				{
					cout << "Failed to launch : hello_world "	<< endl;
				}

				ret = hipModuleUnload(cuModule);

				if (ret != hipSuccess)
				{
					cout << "Failed to unload self fatbin : " << argv[0] << endl;
					return -1;
				}

				if (hipDeviceSynchronize() != hipSuccess)
				{
					printf ("Cuda call failed\n");
				}

				//unmap sutff
				munmap(start_ptr, sz);
				return 0;
			}
		}

	}

	fclose(file);

	return 0;
}

