#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/mman.h>
#include <sys/stat.h>

#include "mriera.h"

using namespace std;

static unsigned long long _find_cubin_offset(ElfW(Shdr) header,
	void* start_ptr, unsigned long long offset, const char* name)
{
	// TODO Parse the ".nv_fatbin" aligning to byte sequence "50 ed 55 ba 01 00 10 00":
	// ...
	// asm(
	// ".section .nv_fatbin, \"a\"\n"
	// ".align 8\n"
	// "fatbinData:\n"
	// ".quad 0x00100001ba55ed50,0x00000000000008a8,0x0000004001010002,0x00000000000007a8\n"
	// ...
	// TODO Find the cubin related to the global method you want to hipModuleGetFunction.
	return offset;
}

int main(int argc, char * argv[])
{
	if (argc != 3)
	{
		printf("%s <elf_filename> <kernel_name>\n", argv[0]);
		return 0;
	}

	hipInit(0);

	// Get number of devices supporting CUDA
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0)
	{
		printf("There is no device supporting CUDA.\n");
		exit(0);
	}
	else
		cout << "Number of devices is "<< deviceCount << endl;

	const char* filename = argv[1];
	const char* kernel_name = argv[2];

	struct stat sb;
	size_t sz = 0;

	// Either Elf64_Ehdr or Elf32_Ehdr depending on architecture.
	ElfW(Ehdr) elf_header;
	ElfW(Shdr) header;

	cout << "opening elf file" << endl;
	FILE* file = fopen(filename, "rb");

	int fd = fileno(file);
	if (fd < 0)
	{
		printf("Could not open file for memory mapping, fd = %i\n", errno);
		exit(1);
	}

	cout << "getting file size" << endl;
	if (fstat(fd, &sb) == -1)					// To obtain file size
		printf("Could not find fstat");
	sz = sb.st_size;

	cout << "Mapping file to memory : " << sz << endl;
	void* start_ptr = mmap(NULL, sz, PROT_READ | PROT_WRITE, MAP_PRIVATE, fd, 0);

	//check if valid elf
	bool b = elf_is_elf64(file);
	fseek(file, 0, SEEK_SET);
	cout << "is ELF file : " << b << endl;
	if (b)
	{
		cout << "Found valid ELF file" << endl;

		unsigned char magic[] = { 0x50, 0xed, 0x55, 0xba, 0x01, 0x00, 0x10, 0x00 };
		size_t cuOffset = (size_t)-1;
		for (size_t i = 0; i < sz - sizeof(magic); i++)
		{
			if (memcmp(start_ptr + i, magic, sizeof(magic)))
				continue;

				cuOffset = i;
				break;
		}
		if (cuOffset == (size_t)-1)
		{
			printf("Could not find the fatbin magic\n");
			exit(1);
		}

		const void * fatbin = &((unsigned char *) start_ptr)[cuOffset];
				
		cout << "fatbin = " << (void*)cuOffset << endl;

		// Get handle for device 0
		hipDevice_t cuDevice;
		hipDeviceGet(&cuDevice, 0);
		// Create context
		hipCtx_t cuContext;
		int ret = hipCtxCreate(&cuContext, 0, cuDevice);
		if (ret != hipSuccess)
			cout << "Could not create context on device 0" << endl;

		// Call hipModuleLoadFatBinary with a base address of the .nv_fatbin + specific cubin offset.
		hipModule_t cuModule;
		ret = hipModuleLoadFatBinary(&cuModule, fatbin);
		if (ret != hipSuccess)
		{
			cout << "Failed to load fatbin : " << filename << " : " << ret << endl;
		}

		hipFunction_t khw;
		ret = hipModuleGetFunction(&khw, cuModule, kernel_name);
		if (ret != hipSuccess)
		{
			cout << "Failed to get " << kernel_name << " from " << filename << " : " << ret << endl;
		}
		else ret = hipModuleLaunchKernel(khw, 1, 1, 1, 1, 1, 1, 0, 0, NULL, 0);

		if (ret != hipSuccess)
		{
			cout << "Failed to launch : " << kernel_name << " : " << ret << endl;
		}

		ret = hipModuleUnload(cuModule);

		if (ret != hipSuccess)
		{
			cout << "Failed to unload self fatbin : " << filename << " : " << ret << endl;
			return -1;
		}

		if (hipDeviceSynchronize() != hipSuccess)
		{
			printf ("Cuda call failed\n");
		}

		//unmap sutff
		munmap(start_ptr, sz);
		return 0;
	}

	fclose(file);

	return 0;
}

