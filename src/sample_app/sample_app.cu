
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ void hello_world()
{
	printf("Hello, world!\n");
}

int main(int argc, char* argv[])
{
	hello_world<<<1, 1>>>();
	hipDeviceSynchronize();
	return 0;
}
