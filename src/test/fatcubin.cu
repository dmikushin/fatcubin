#include "fatcubin.h"

#include <cstdio>
#include <hip/hip_runtime.h>

int main(int argc, char * argv[])
{
	if (argc != 3)
	{
		printf("%s <elf_filename> <kernel_name>\n", argv[0]);
		return 0;
	}

	int ret = hipInit(0);
	if (ret != hipSuccess)
	{
		fprintf(stderr, "Could not initialize the CUDA driver\n");
		return -1;
	}

	// Get number of devices supporting CUDA
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0)
	{
		fprintf(stderr, "There are no devices supporting CUDA\n");
		return -1;
	}

	// Get handle for device 0
	hipDevice_t cuDevice;
	hipDeviceGet(&cuDevice, 0);

	// Create context
	hipCtx_t cuContext;
	ret = hipCtxCreate(&cuContext, 0, cuDevice);
	if (ret != hipSuccess)
	{
		fprintf(stderr, "Could not create context on device 0\n");
		return -1;
	}

	const char* filename = argv[1];
	const char* kernel_name = argv[2];

	FatCubin fatCubin(filename);
	
	if (!fatCubin.is_valid())
	{
		fprintf(stderr, "Not a valid ELF file: \"%s\"\n", filename);
		return -1;
	}

	std::vector<void*> cubins;
	fatCubin.getAll(cubins);
	if (!cubins.size())
	{
		fprintf(stderr, "Could not find any CUBINs in file \"%s\"\n", filename);
		return -1;
	}	

	bool found = false;
	hipFunction_t cuFunction;
	for (auto cubin : cubins)
	{
		hipModule_t cuModule;
		ret = hipModuleLoadFatBinary(&cuModule, cubin);
		if (ret != hipSuccess)
		{
			fprintf(stderr, "Failed to load module from %p : errno = %d\n", filename, ret);
			continue;
		}
		
		ret = hipModuleGetFunction(&cuFunction, cuModule, kernel_name);
		if (ret == hipSuccess)
		{
			found = true;
			break;
		}
		
		hipModuleUnload(cuModule);
	}

	if (!found)
	{
		fprintf(stderr, "Failed to get \"%s\" from \"%s\"\n", kernel_name, filename);
		return -1;
	}

	ret = hipModuleLaunchKernel(cuFunction, 1, 1, 1, 1, 1, 1, 0, 0, NULL, 0);
	if (ret != hipSuccess)
	{
		fprintf(stderr, "Failed to launch \"%s\" : errno = %d\n", kernel_name, ret);
		return -1;
	}

	ret = hipCtxSynchronize();
	if (ret != hipSuccess)
	{
		fprintf(stderr, "CUDA kernel \"%s\" launch failed : errno = %d\n", kernel_name, ret);
		return -1;
	}

	return 0;
}

